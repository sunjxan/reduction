#include "hip/hip_runtime.h"
#include "common.hpp"

__global__ void kernel(const real *A, size_t size, real *B, size_t group_count, size_t group_size)
{
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < group_count) {
        unsigned beg = idx * group_size, end = beg + group_size;
        if (end > size) {
            end = size;
        }
        real sum = 0.0;
        for (size_t i = beg; i < end; ++i) {
            sum += A[i];
        }
        B[idx] = sum;
    }
}

void reduce(const real *A, size_t size, real *result)
{
    const size_t group_count = 1e6, group_size = DIVUP(size, group_count), total_size = group_count * real_size;

    real *B, *h_B;
    CHECK(hipMalloc(&B, total_size));
    CHECK(hipHostMalloc(&h_B, total_size));

    unsigned block_size = 128, grid_size = DIVUP(group_count, block_size);
    kernel<<<grid_size, block_size>>>(A, size, B, group_count, group_size);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_B, B, total_size, hipMemcpyDeviceToHost));

    real sum = 0.0;
    for (size_t i = 0; i < group_count; ++i) {
        sum += h_B[i];
    }
    *result = sum;

    CHECK(hipHostFree(h_B));
    CHECK(hipFree(B));
}

int main()
{
    launch_gpu();
    return 0;
}