#include "hip/hip_runtime.h"
#include "common.hpp"

__global__ void kernel(const real *A, size_t size, real *B, size_t group_count)
{
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < group_count) {
        real sum = 0.0;
        for (size_t i = idx; i < size; i += group_count) {
            sum += A[i];
        }
        B[idx] = sum;
    }
}

void reduce(const real *d_A, size_t size, real *h_result)
{
    const size_t group_count = 1 << 20, total_size = group_count * real_size;

    real *d_B = nullptr, *h_B = nullptr;
    CHECK(hipMalloc(&d_B, total_size));
    CHECK(hipHostMalloc(&h_B, total_size));

    unsigned block_size = 1024, grid_size = DIVUP(group_count, block_size);
    kernel<<<grid_size, block_size>>>(d_A, size, d_B, group_count);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_B, d_B, total_size, hipMemcpyDeviceToHost));

    real sum = 0.0;
    for (size_t i = 0; i < group_count; ++i) {
        sum += h_B[i];
    }
    *h_result = sum;

    CHECK(hipHostFree(h_B));
    CHECK(hipFree(d_B));
}

int main()
{
    launch_gpu();
    return 0;
}