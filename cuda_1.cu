#include "hip/hip_runtime.h"
#include "common.hpp"

__global__ void kernel(const real *A, size_t size, real *result)
{
    real sum = 0.0;
    for (size_t i = 0; i < size; ++i) {
        sum += A[i];
    }
    *result = sum;
}

void reduce(const real *d_A, size_t size, real *h_result)
{
    real *d_result = nullptr;
    CHECK(hipMalloc(&d_result, real_size));

    kernel<<<1, 1>>>(d_A, size, d_result);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_result, d_result, real_size, hipMemcpyDeviceToHost));
    CHECK(hipFree(d_result));
}

int main()
{
    launch_gpu();
    return 0;
}