#include "hip/hip_runtime.h"
#include "common.hpp"

__global__ void kernel(const real *A, size_t size, real *result)
{
    real sum = 0.0;
    for (size_t i = 0; i < size; ++i) {
        sum += A[i];
    }
    *result = sum;
}

void reduce(const real *A, size_t size, real *result)
{
    real *d_result;
    CHECK(hipMalloc(&d_result, real_size));

    kernel<<<1, 1>>>(A, size, d_result);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(result, d_result, real_size, hipMemcpyDeviceToHost));
    CHECK(hipFree(d_result));
}

int main()
{
    launch_gpu();
    return 0;
}