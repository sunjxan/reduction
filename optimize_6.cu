#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

#include "common.hpp"

using namespace cooperative_groups;

// 使用协作组

__global__ void kernel(const real *A, size_t size, real *B)
{
    unsigned tid = threadIdx.x, bid = blockIdx.x, bdx = blockDim.x, idx = bid * bdx + tid;
    extern __shared__ real s_a[];
    if (idx >= size) {
        s_a[tid] = 0.0;
        return;
    }

    size_t pos = idx, thread_count = gridDim.x * blockDim.x;
    real v = A[pos];
    while (pos + thread_count < size) {
        pos += thread_count;
        v += A[pos];
    }
    s_a[tid] = v;
    __syncthreads();

    for (size_t stride = bdx >> 1; stride > 32; stride >>= 1) {
        if (tid < stride) {
            s_a[tid] += s_a[tid + stride];
        }
        __syncthreads();
    }

    if (tid < 32) {
        s_a[tid] += s_a[tid + 32];
        __syncwarp();

        v = s_a[tid];
        thread_block_tile<32> g = tiled_partition<32>(this_thread_block());
        v += g.shfl_down(v, 16);
        v += g.shfl_down(v, 8);
        v += g.shfl_down(v, 4);
        v += g.shfl_down(v, 2);
        v += g.shfl_down(v, 1);
    }

    if (!tid) {
        B[bid] = v;
    }
}

void reduce(const real *d_A, size_t size, real *h_result)
{
    // 以1:times的比例估算需要的线程数groups，block_size应是2的整数幂
    unsigned times = 10, groups = DIVUP(size, times), block_size = 1024, grid_size = DIVUP(groups, block_size);
    size_t B_size = grid_size * real_size;

    real *d_B = nullptr;
    CHECK(hipMalloc(&d_B, B_size));

    kernel<<<grid_size, block_size, block_size * real_size>>>(d_A, size, d_B);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    // 保证grid_size=1,block_size=1024的kernel能完成全部计算
    real *d_result = nullptr;
    CHECK(hipMalloc(&d_result, real_size));
    CHECK(hipMemset(d_result, 0, real_size));

    kernel<<<1, block_size, block_size * real_size>>>(d_B, grid_size, d_result);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_result, d_result, real_size, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_result));
    CHECK(hipFree(d_B));
}

int main()
{
    launch_gpu();
    return 0;
}
