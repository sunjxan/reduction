#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

#include "common.hpp"

using namespace cooperative_groups;

// 使用协作组

__global__ void kernel(const real *A, size_t size, real *B)
{
    unsigned tid = threadIdx.x, bid = blockIdx.x, bdx = blockDim.x, idx = bid * bdx + tid;
    extern __shared__ real s_a[];
    if (idx >= size) {
        s_a[tid] = 0.0;
        return;
    }

    size_t pos = idx, thread_count = gridDim.x * blockDim.x;
    real v = A[pos];
    while (pos + thread_count < size) {
        pos += thread_count;
        v += A[pos];
    }
    s_a[tid] = v;
    __syncthreads();

    for (size_t stride = bdx >> 1; stride > 32; stride >>= 1) {
        if (tid < stride) {
            s_a[tid] += s_a[tid + stride];
        }
        __syncthreads();
    }

    if (tid < 32) {
        s_a[tid] += s_a[tid + 32];

        v = s_a[tid];
        thread_block_tile<32> g = tiled_partition<32>(this_thread_block());
        for (size_t stride = 16; stride > 0; stride >>= 1) {
            v += g.shfl_down(v, stride);
        }
    }

    if (!tid) {
        B[bid] = v;
    }
}

void reduce(const real *d_A, size_t size, real *h_result)
{
    // 以1:times的比例估算需要的线程数groups，block_size应是2的整数幂，且大于等于64
    unsigned times = 10, groups = DIVUP(size, times), block_size = 1024, grid_size = DIVUP(groups, block_size);
    size_t B_size = grid_size * real_size;

    real *d_B = nullptr;
    CHECK(hipMalloc(&d_B, B_size));

    kernel<<<grid_size, block_size, block_size * real_size>>>(d_A, size, d_B);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    unsigned block_size2 = 512;
    kernel<<<1, block_size2, block_size2 * real_size>>>(d_B, grid_size, d_B);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_result, d_B, real_size, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_B));
}

int main()
{
    launch_gpu();
    return 0;
}
