#include "hip/hip_runtime.h"
#include "common.hpp"

// 为了代替最后传回host计算的步骤，调用两次kernel函数，直接归约成一个数
// 受限于block_size<=1024，每个线程折半之前先累加times倍范围的元素
// 为了第二个kernel可以一个block完成计算，需要调整times值

__global__ void kernel(const real *A, size_t size, real *B, size_t thread_count, real *C, unsigned times)
{
    unsigned tid = threadIdx.x, bid = blockIdx.x, bdx = blockDim.x, idx = bid * bdx + tid;
    if (idx >= size) {
        return;
    }

    real v = A[idx];
    for (size_t i = 1; i < times; ++i) {
        if (idx + i * thread_count < size) {
            v += A[idx + i * thread_count];
        } else {
            break;
        }
    }
    B[idx] = v;
    __syncthreads();

    real *Bx = B + bid * bdx;
    for (size_t stride = bdx >> 1; stride > 0; stride >>= 1) {
        if (tid < stride) {
            Bx[tid] += Bx[tid + stride];
        }
        __syncthreads();
    }

    if (!tid) {
        C[bid] = Bx[0];
    }
}

void reduce(const real *d_A, size_t size, real *h_result)
{
    if (!size) {
        *h_result = 0.0;
        return;
    }

    // 以1:times的比例估算数组B的长度和需要的线程数groups，block_size应是2的整数幂，thread_count是实际使用的线程数
    unsigned times = 10, groups = DIVUP(size, times), block_size = 1024, grid_size = DIVUP(groups, block_size);
    size_t thread_count = grid_size * block_size, B_size = thread_count * real_size, C_size = grid_size * real_size;

    real *d_B = nullptr;
    CHECK(hipMalloc(&d_B, B_size));
    // 为折半设置初值0
    CHECK(hipMemset(d_B, 0, B_size));

    real *d_C = nullptr, *h_C = nullptr;
    CHECK(hipMalloc(&d_C, C_size));
    CHECK(hipHostMalloc(&h_C, C_size));

    kernel<<<grid_size, block_size>>>(d_A, size, d_B, thread_count, d_C, times);
    CHECK(hipDeviceSynchronize());

    // 保证grid_size=1,block_size=1024的kernel能完成全部计算
    times = DIVUP(grid_size, block_size);
    real *d_result = nullptr;
    CHECK(hipMalloc(&d_result, real_size));
    
    // 为折半设置初值0
    CHECK(hipMemset(d_B, 0, B_size));

    kernel<<<1, block_size>>>(d_C, grid_size, d_B, block_size, d_result, times);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_result, d_result, real_size, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_result));
    CHECK(hipHostFree(h_C));
    CHECK(hipFree(d_C));
    CHECK(hipFree(d_B));
}

int main()
{
    launch_gpu();
    return 0;
}
