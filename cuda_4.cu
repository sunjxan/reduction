#include "hip/hip_runtime.h"
#include "common.hpp"

// 每个block计算一个部分和，相邻配对

__global__ void kernel(const real *A, size_t size, real *B, size_t thread_count, real *C)
{
    unsigned tid = threadIdx.x, bid = blockIdx.x, bdx = blockDim.x, idx = bid * bdx + tid;
    if (idx >= thread_count) {
        return;
    }

    size_t pos = idx << 1;
    real v = A[pos];
    if (pos + 1 < size) {
        v += A[pos + 1];
    }
    B[idx] = v;
    __syncthreads();

    real *Bx = B + bid * bdx;
    for (size_t stride = 1; stride < bdx; ) {
        size_t next_stride = stride << 1;
        // 添加边界检查
        if (!(tid % next_stride) && tid + stride < bdx && idx + stride < thread_count) {
            Bx[tid] += Bx[tid + stride];
        }
        stride = next_stride;
        __syncthreads();
    }

    if (!tid) {
        C[bid] = Bx[0];
    }
}

void reduce(const real *d_A, size_t size, real *h_result)
{
    size_t thread_count = (size + 1) >> 1;
    real *d_B = nullptr;
    CHECK(hipMalloc(&d_B, thread_count * real_size));

    unsigned block_size = 1024, grid_size = DIVUP(thread_count, block_size);
    size_t total_size = grid_size * real_size;
    real *d_C = nullptr, *h_C = nullptr;
    CHECK(hipMalloc(&d_C, total_size));
    CHECK(hipHostMalloc(&h_C, total_size));

    kernel<<<grid_size, block_size>>>(d_A, size, d_B, thread_count, d_C);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_C, d_C, total_size, hipMemcpyDeviceToHost));
    real sum = 0.0;
    for (size_t i = 0; i < grid_size; ++i) {
        sum += h_C[i];
    }
    *h_result = sum;

    CHECK(hipHostFree(h_C));
    CHECK(hipFree(d_C));
    CHECK(hipFree(d_B));
}

int main()
{
    launch_gpu();
    return 0;
}